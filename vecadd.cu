
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>
#include <ctime>
#include <chrono>

// CUDA kernel for vector addition
__global__
void vectorAddition(const int* A, const int* B, int* C, int size)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < size)
    {
        C[index] = A[index] + B[index];
    }
}

// Function to perform vector addition sequentially
void sequentialVectorAddition(const int* A, const int* B, int* C, int size)
{
    for (int i = 0; i < size; ++i)
    {
        C[i] = A[i] + B[i];
    }
}

int main()
{
    const int size = 1000000;  // Size of the vectors

    // Allocate memory for vectors on host
    int* A = new int[size];
    int* B = new int[size];
    int* C = new int[size];

    // Initialize vectors with random values
    srand(static_cast<unsigned>(time(0)));
    for (int i = 0; i < size; ++i)
    {
        A[i] = rand() % 100;
        B[i] = rand() % 100;
    }

    // Allocate memory for vectors on device
    int* d_A, * d_B, * d_C;
    hipMalloc((void**)&d_A, sizeof(int) * size);
    hipMalloc((void**)&d_B, sizeof(int) * size);
    hipMalloc((void**)&d_C, sizeof(int) * size);

    // Copy input vectors from host to device
    hipMemcpy(d_A, A, sizeof(int) * size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, sizeof(int) * size, hipMemcpyHostToDevice);

    // Set up thread configuration
    int threadsPerBlock = 256;
    int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;

    // Start timer for parallel algorithm
    auto startParallel = std::chrono::high_resolution_clock::now();

    // Launch kernel for vector addition in parallel
    vectorAddition<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, size);

    // Wait for kernel to finish execution
    hipDeviceSynchronize();

    // End timer for parallel algorithm
    auto endParallel = std::chrono::high_resolution_clock::now();

    // Copy result vector from device to host
    hipMemcpy(C, d_C, sizeof(int) * size, hipMemcpyDeviceToHost);

    // Start timer for sequential algorithm
    auto startSequential = std::chrono::high_resolution_clock::now();

    // Perform vector addition sequentially
    sequentialVectorAddition(A, B, C, size);

    // End timer for sequential algorithm
    auto endSequential = std::chrono::high_resolution_clock::now();

    // Calculate elapsed time for parallel algorithm
    auto durationParallel = std::chrono::duration_cast<std::chrono::microseconds>(endParallel - startParallel);

    // Calculate elapsed time for sequential algorithm
    auto durationSequential = std::chrono::duration_cast<std::chrono::microseconds>(endSequential - startSequential);

    // Print performance results
    std::cout << "Parallel Algorithm Time: " << durationParallel.count() << " microseconds" << std::endl;
    std::cout << "Sequential Algorithm Time: " << durationSequential.count() << " microseconds" << std::endl;

    // Print the results of vector addition (first 10 elements)
    std::cout << "Vector Addition Result:" << std::endl;
    for (int i = 0; i < 10; ++i)
    {
        std::cout << A[i] << " + " << B[i] << " = " << C[i] << std::endl;
    }

    // Free memory
    delete[] A;
    delete[] B;
    delete[] C;
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}

